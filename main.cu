
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_cuda() { printf("Hello, CUDA!\n"); }

int main() {
  // Launch the kernel
  hello_cuda<<<1, 1>>>();
  hipDeviceSynchronize();

  std::cout << "Hello, World!" << std::endl;

  return 0;
}
